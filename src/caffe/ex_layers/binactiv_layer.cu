#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/ex_layers/binactiv_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

#define sign(x) ((x)>=0?1:-1)

template <typename Dtype>
__global__ void BinActiv_FW_Kernal(const int count, const int C, const int map_size, const Dtype* bottom_data, Dtype* sumA, Dtype* signA) {
  CUDA_KERNEL_LOOP(index, count) {
    const int batch_id = index / map_size;
    const int coordinate = index % map_size;
    sumA[index] = 0;
    for (int _c = 0; _c < C; _c++) {
      sumA[index] += bottom_data[ batch_id*C*map_size + _c*map_size + coordinate ] / C;
      signA[ batch_id*C*map_size + _c*map_size + coordinate ] =
        sign( bottom_data[ batch_id*C*map_size + _c*map_size + coordinate ] );
    }
  }
}

template <typename Dtype>
__global__ void BinActiv_Sign_Kernal(const int count, const int C, const int map_size, const Dtype* bottom_data, Dtype* signA) {
  CUDA_KERNEL_LOOP(index, count) {
    const int batch_id = index / map_size;
    const int coordinate = index % map_size;
    for (int _c = 0; _c < C; _c++) {
      signA[ batch_id*C*map_size + _c*map_size + coordinate ] =
        sign( bottom_data[ batch_id*C*map_size + _c*map_size + coordinate ] );
    }
  }
}

template <typename Dtype>
void BinActivLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  DLOG(INFO) << "-----> " << this->layer_param_.name() << ",,, Forward_gpu start ";
  const Dtype* bottom_data = bottom[0]->gpu_data();
  if (!no_k_){
    Dtype* sumA = convolution_bottom_vec_[0]->mutable_gpu_data();
    Dtype* signA = top[0]->mutable_gpu_data();
    const int C = bottom[0]->channels();
    const int count = convolution_bottom_vec_[0]->count();
    const int map_size = bottom[0]->height()*bottom[0]->width();
    DLOG(INFO) << "-----> " << this->layer_param_.name() << ",,, Forward_gpu C : " << C << ", count : " << count << ", map_size : " << map_size;
    DLOG(INFO) << "-----> " << this->layer_param_.name() << ",,, signA : " << top[0]->num() << ", " << top[0]->channels() << ", " << top[0]->height() << ", " << top[0]->width();
    BinActiv_FW_Kernal<Dtype>
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, C, map_size, bottom_data, sumA, signA);

    DLOG(INFO) << "-----> " << this->layer_param_.name() << ",,, BinActiv_FW_Kernal Done ";
    CUDA_POST_KERNEL_CHECK;
    //hipDeviceSynchronize();
    convolution_layer_->Forward(convolution_bottom_vec_, convolution_top_vec_);

    DLOG(INFO) << "-----> " << this->layer_param_.name() << ",,, convolution_layer Forward";
    const int size_kernal = this->layer_param_.convolution_param().kernel_size(0)
            * this->layer_param_.convolution_param().kernel_size(0);
    CHECK_EQ(top[1]->count(), convolution_top_vec_[0]->count());
    caffe_copy(top[1]->count(), convolution_top_vec_[0]->gpu_data(), top[1]->mutable_gpu_data());
    caffe_gpu_scale(top[1]->count(), Dtype(1)/size_kernal, top[1]->gpu_data(),  top[1]->mutable_gpu_data());
    DLOG(INFO) << "-----> " << this->layer_param_.name() << ",,, Forward_gpu left ";
  }
  else{
    Dtype* signA = top[0]->mutable_gpu_data();
    const int C = bottom[0]->channels();
    /*const int count = convolution_bottom_vec_[0]->count();*/
    const int count = bottom[0]->num()*bottom[0]->height()*bottom[0]->width();
    const int map_size = bottom[0]->height()*bottom[0]->width();
    BinActiv_Sign_Kernal<Dtype>
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, C, map_size, bottom_data, signA);
  }
}

template <typename Dtype>
__global__ void BinActiv_BP_Kernal(const int count, const Dtype* bottom_data, const Dtype* top_diff, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, count) {
    if ( std::abs(bottom_data[index]) <= Dtype(1) ) {
      bottom_diff[ index ] = top_diff[ index ];
    } else {
      bottom_diff[ index ] = Dtype(0);
    }
  }
}

template <typename Dtype>
void BinActivLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if ( propagate_down[0] == false ) return;
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = top[0]->count();
  BinActiv_BP_Kernal<Dtype>
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_diff, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(BinActivLayer);

}  // namespace caffe
