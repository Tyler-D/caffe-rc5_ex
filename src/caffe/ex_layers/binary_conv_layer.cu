#include "hip/hip_runtime.h"
#ifdef USE_CUDNN
#include <vector>

#include "caffe/ex_layers/binary_conv_layer.hpp"

namespace caffe {

#define sign(x) ((x)>=0?1:-1)
__global__ void binary_sync_conv_groups() { }

template <typename Dtype>
__global__ void BinaryGPU_binarize(const int n, const int num, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype sum = 0;
    for (int coor = 0; coor < num; coor++) {
      sum += std::abs(in[index * num + coor]) / Dtype(num);
    }
    for (int coor = 0; coor < num; coor++) {
      out[index * num + coor] = sign(in[index * num + coor]) * sum;
    }
  }  
}

template <typename Dtype>
void BinaryConvolutionLayer<Dtype>::binarizeGPUTo(const shared_ptr<Blob<Dtype> > weights, const shared_ptr<Blob<Dtype> > wb) {
  const int count = weights->num();
  const int div = weights->count() / count;
  BinaryGPU_binarize<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, div, weights->gpu_data(), wb->mutable_gpu_data() );
}

template <typename Dtype>
void BinaryConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  binarizeGPUTo(this->blobs_[0], W_b);
  copyGPUTo(this->blobs_[0], W_buffer);
  copyGPUTo(W_b, this->blobs_[0]);
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(hipdnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
#if CUDNN_VERSION_MIN(4, 0, 0)
        CUDNN_CHECK(hipdnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
#else
        CUDNN_CHECK(hipdnnAddTensor(handle_[g], CUDNN_ADD_SAME_C,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
#endif
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    binary_sync_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void BinaryConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(hipdnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(hipdnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(hipdnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    binary_sync_conv_groups<<<1, 1>>>();
  }
  copyGPUTo(W_buffer, this->blobs_[0]);
}

template void BinaryConvolutionLayer<float>::binarizeGPUTo(const shared_ptr<Blob<float> > weights, const shared_ptr<Blob<float> > wb);
template void BinaryConvolutionLayer<double>::binarizeGPUTo(const shared_ptr<Blob<double> > weights, const shared_ptr<Blob<double> > wb);

INSTANTIATE_LAYER_GPU_FUNCS(BinaryConvolutionLayer);

}  // namespace caffe
#else

#include "caffe/ex_layers/binary_conv_layer.hpp"

namespace caffe {

template <typename Dtype>
void BinaryConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  Forward_cpu(bottom, top);
}

template <typename Dtype>
void BinaryConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(BinaryConvolutionLayer);

}
#endif
